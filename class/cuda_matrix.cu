#include <hip/hip_runtime.h>

#include <stdio.h>

// Function to check CUDA errors
static void cuda_error(const char *msg) {
  hipError_t err = hipGetLastError();
  if (err != hipSuccess) {
    fprintf(stderr, "CUDA error: %s (%s)\n", msg, hipGetErrorName(err));
    exit(1);
  }
}

__global__ void matrixMul(float *A, float *B, float *C, int n) {
  int row = blockIdx.y * blockDim.y + threadIdx.y;
  int col = blockIdx.x * blockDim.x + threadIdx.x;

  float sum = 0.0f;
  if (row < n && col < n) {
    for (int k = 0; k < n; k++) {
      sum += A[row * n + k] * B[k * n + col];
    }
    C[row * n + col] = sum;
  }
}

int main(int argc, char *argv[]) {
  // Matrix size (modify as needed)
  int n = 1024;

  // Allocate host memory for matrices
  float *h_A, *h_B, *h_C;
  h_A = (float *)malloc(n * n * sizeof(float));
  h_B = (float *)malloc(n * n * sizeof(float));
  h_C = (float *)malloc(n * n * sizeof(float));

  // Initialize matrices (replace with your initialization logic)
  for (int i = 0; i < n * n; i++) {
    h_A[i] = 1.0f; // Modify as needed
    h_B[i] = 2.0f; // Modify as needed
  }

  // Allocate device memory for matrices
  float *d_A, *d_B, *d_C;
  cuda_error("malloc d_A");
  hipMalloc((void **)&d_A, n * n * sizeof(float));
  cuda_error("malloc d_B");
  hipMalloc((void **)&d_B, n * n * sizeof(float));
  cuda_error("malloc d_C");
  hipMalloc((void **)&d_C, n * n * sizeof(float));

  // Copy matrices from host to device
  cuda_error("memcpy d_A");
  hipMemcpy(d_A, h_A, n * n * sizeof(float), hipMemcpyHostToDevice);
  cuda_error("memcpy d_B");
  hipMemcpy(d_B, h_B, n * n * sizeof(float), hipMemcpyHostToDevice);

  // Launch kernel
  int threadsPerBlock = 256;
  int blocksPerGrid = (n + threadsPerBlock - 1) / threadsPerBlock;
  cuda_error("launch kernel");
  matrixMul<<<blocksPerGrid, threadsPerBlock>>>(d_A, d_B, d_C, n);

  // Wait for kernel to finish execution
  hipDeviceSynchronize();

  // Copy result from device to host
  cuda_error("memcpy h_C");
  hipMemcpy(h_C, d_C, n * n * sizeof(float), hipMemcpyDeviceToHost);

  // Free device memory
  hipFree(d_A);
  hipFree(d_B);
  hipFree(d_C);

  // Print some elements of the resulting matrix (optional)
  for (int i = 0; i < 4; i++) {
    for (int j = 0; j < 4; j++) {
      printf("%.2f ", h_C[i * n + j]);
    }
    printf("\n");
  }

  // Free host memory
  free(h_A);
  free(h_B);
  free(h_C);

  return 0;
}
